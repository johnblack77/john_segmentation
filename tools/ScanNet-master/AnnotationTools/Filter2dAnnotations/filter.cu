#include "hip/hip_runtime.h"
#ifndef _FILTER_
#define _FILTER_

#include <cutil_inline.h>
#include <cutil_math.h>
#include "GlobalDefines.h"
#include "cuda_SimpleMatrixUtil.h"


#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Convert Float4 Color to UCHAR4
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void convertColorFloat4ToUCHAR4Device(uchar4* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	float4 color = d_input[y*width + x];
	d_output[y*width + x] = make_uchar4(color.x*255.0f, color.y*255.0f, color.z*255.0f, color.w*255.0f);
}

extern "C" void convertColorFloat4ToUCHAR4(uchar4* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 blockSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 gridSize(T_PER_BLOCK, T_PER_BLOCK);

	convertColorFloat4ToUCHAR4Device << <blockSize, gridSize >> >(d_output, d_input, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Convert Color to Intensity
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void convertColorToIntensityFloatDevice(float* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	const float4 color = d_input[y*width + x];
	d_output[y*width + x] = 0.299f*color.x + 0.587f*color.y + 0.114f*color.z;
}

extern "C" void convertColorToIntensityFloat(float* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertColorToIntensityFloatDevice << <gridSize, blockSize >> >(d_output, d_input, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Convert Depth to Camera Space Positions
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void convertDepthFloatToCameraSpaceFloat4Device(float4* d_output, float* d_input, float4x4 intrinsicsInv, 
	unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		d_output[y*width+x] = make_float4(MINF, MINF, MINF, MINF);

		float depth = d_input[y*width+x];

		if(depth != MINF)
		{
			float4 cameraSpace(intrinsicsInv*make_float4((float)x*depth, (float)y*depth, depth, depth));
			d_output[y*width+x] = make_float4(cameraSpace.x, cameraSpace.y, cameraSpace.w, 1.0f);
		}
	}
}

extern "C" void convertDepthFloatToCameraSpaceFloat4(float4* d_output, float* d_input, float4x4 intrinsicsInv, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertDepthFloatToCameraSpaceFloat4Device<<<gridSize, blockSize>>>(d_output, d_input, intrinsicsInv, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Compute Normal Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void computeNormalsDevice(float4* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= width || y >= height) return;

	d_output[y*width+x] = make_float4(MINF, MINF, MINF, MINF);

	if(x > 0 && x < width-1 && y > 0 && y < height-1)
	{
		const float4 CC = d_input[(y+0)*width+(x+0)];
		const float4 PC = d_input[(y+1)*width+(x+0)];
		const float4 CP = d_input[(y+0)*width+(x+1)];
		const float4 MC = d_input[(y-1)*width+(x+0)];
		const float4 CM = d_input[(y+0)*width+(x-1)];

		if(CC.x != MINF && PC.x != MINF && CP.x != MINF && MC.x != MINF && CM.x != MINF)
		{
			const float3 n = cross(make_float3(PC)-make_float3(MC), make_float3(CP)-make_float3(CM));
			const float  l = length(n);

			if(l > 0.0f)
			{
				d_output[y*width+x] = make_float4(n/-l, 1.0f);
			}
		}
	}
}

extern "C" void computeNormals(float4* d_output, float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1)/T_PER_BLOCK, (height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeNormalsDevice<<<gridSize, blockSize>>>(d_output, d_input, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

/////////////////////////////////////////////
// Transform
/////////////////////////////////////////////

__global__ void transformNormalMapDevice(float4* d_normals, unsigned int imageWidth, unsigned int imageHeight, float4x4 transform)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	const int index = y*imageWidth+x;

	if(x >= 0 && x < imageWidth && y >= 0 && y < imageHeight)
	{
		if(d_normals[index].x != MINF)
		{
			float3 n = transform.getFloat3x3() * make_float3(d_normals[index].x,d_normals[index].y,d_normals[index].z); 
			d_normals[index] = make_float4(n, 0.0f);
		}
	}
}

extern "C" void transformNormalMap(float4* d_normals, unsigned int imageWidth, unsigned int imageHeight, float4x4 transform)
{
	const dim3 gridSize((imageWidth + T_PER_BLOCK - 1)/T_PER_BLOCK, (imageHeight + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	transformNormalMapDevice<<<gridSize, blockSize>>>(d_normals, imageWidth, imageHeight, transform);
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Bilateral Filter Float Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

inline __device__ float gaussR(float sigma, float dist)
{
	return exp(-(dist*dist) / (2.0*sigma*sigma));
}

inline __device__ float linearR(float sigma, float dist)
{
	return max(1.0f, min(0.0f, 1.0f - (dist*dist) / (2.0*sigma*sigma)));
}

inline __device__ float gaussD(float sigma, int x, int y)
{
	return exp(-((x*x + y*y) / (2.0f*sigma*sigma)));
}

inline __device__ float gaussD(float sigma, int x)
{
	return exp(-((x*x) / (2.0f*sigma*sigma)));
}

__global__ void bilateralFilterFloatMapDevice(float* d_output, float* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	d_output[y*width + x] = MINF;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	const float depthCenter = d_input[y*width + x];
	if (depthCenter != MINF)
	{
		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const float currentDepth = d_input[n*width + m];

					if (currentDepth != MINF) {
						const float weight = gaussD(sigmaD, m - x, n - y)*gaussR(sigmaR, currentDepth - depthCenter);

						sumWeight += weight;
						sum += weight*currentDepth;
					}
				}
			}
		}

		if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
	}
}

extern "C" void bilateralFilterFloatMap(float* d_output, float* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	bilateralFilterFloatMapDevice << <gridSize, blockSize >> >(d_output, d_input, sigmaD, sigmaR, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Bilateral Filter Float4 Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void bilateralFilterFloat4MapDevice(float4* d_output, float4* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	//d_output[y*width+x] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	d_output[y*width + x] = make_float4(MINF, MINF, MINF, MINF);

	float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	float sumWeight = 0.0f;

	const float4 depthCenter = d_input[y*width + x];
	if (depthCenter.x != MINF) {
		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const float4 currentDepth = d_input[n*width + m];

					if (currentDepth.x != MINF) {
						const float weight = gaussD(sigmaD, m - x, n - y)*gaussR(sigmaR, length(currentDepth - depthCenter));

						sum += weight*currentDepth;
						sumWeight += weight;
					}
				}
			}
		}
	}
	if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
}

extern "C" void bilateralFilterFloat4Map(float4* d_output, float4* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	bilateralFilterFloat4MapDevice << <gridSize, blockSize >> >(d_output, d_input, sigmaD, sigmaR, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Gauss Filter Float Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void gaussFilterFloatMapDevice(float* d_output, float* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	d_output[y*width + x] = MINF;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	const float depthCenter = d_input[y*width + x];
	if (depthCenter != MINF)
	{
		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const float currentDepth = d_input[n*width + m];

					if (currentDepth != MINF && fabs(depthCenter - currentDepth) < sigmaR)
					{
						const float weight = gaussD(sigmaD, m - x, n - y);

						sumWeight += weight;
						sum += weight*currentDepth;
					}
				}
			}
		}
	}

	if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
}

extern "C" void gaussFilterFloatMap(float* d_output, float* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	gaussFilterFloatMapDevice << <gridSize, blockSize >> >(d_output, d_input, sigmaD, sigmaR, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Gauss Filter Float4 Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void gaussFilterFloat4MapDevice(float4* d_output, float4* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	//d_output[y*width+x] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	d_output[y*width + x] = make_float4(MINF, MINF, MINF, MINF);

	float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	float sumWeight = 0.0f;

	const float4 center = d_input[y*width + x];
	if (center.x != MINF) {
		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const float4 current = d_input[n*width + m];

					if (current.x != MINF) {
						if (length(center - current) < sigmaR)
						{
							const float weight = gaussD(sigmaD, m - x, n - y);

							sumWeight += weight;
							sum += weight*current;
						}
					}
				}
			}
		}
	}

	if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
}

extern "C" void gaussFilterFloat4Map(float4* d_output, float4* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	gaussFilterFloat4MapDevice << <gridSize, blockSize >> >(d_output, d_input, sigmaD, sigmaR, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Compute Edge Mask
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void computeMaskEdgeMapFloat4Device(unsigned char* d_output, float4* d_input, float* d_indepth, float threshold, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	d_output[y*width + x] = 1;
	d_output[width*height + y*width + x] = 1;

	const float thre = threshold *threshold *3.0f;
	if (x > 0 && y > 0 && x < width - 1 && y < height - 1)
	{
		if (d_indepth[y*width + x] == MINF)
		{
			d_output[y*width + x] = 0;
			d_output[y*width + x - 1] = 0;
			d_output[width*height + y*width + x] = 0;
			d_output[width*height + (y - 1)*width + x] = 0;

			return;
		}

		const float4& p0 = d_input[(y + 0)*width + (x + 0)];
		const float4& p1 = d_input[(y + 0)*width + (x + 1)];
		const float4& p2 = d_input[(y + 1)*width + (x + 0)];

		float dU = sqrt(((p1.x - p0.x)*(p1.x - p0.x) + (p1.y - p0.y) * (p1.y - p0.y) + (p1.z - p0.z)*(p1.z - p0.z)) / 3.0f);
		float dV = sqrt(((p2.x - p0.x)*(p2.x - p0.x) + (p2.y - p0.y) * (p2.y - p0.y) + (p2.z - p0.z)*(p2.z - p0.z)) / 3.0f);

		//float dgradx = abs(d_indepth[y*width+x-1] + d_indepth[y*width+x+1] - 2.0f * d_indepth[y*width+x]);
		//float dgrady = abs(d_indepth[y*width+x-width] + d_indepth[y*width+x+width] - 2.0f * d_indepth[y*width+x]);


		if (dU > thre) d_output[y*width + x] = 0;
		if (dV > thre) d_output[width*height + y*width + x] = 0;

		//remove depth discontinuities
		const int r = 1;
		const float thres = 0.01f;

		const float pCC = d_indepth[y*width + x];
		for (int i = -r; i <= r; i++)
		{
			for (int j = -r; j <= r; j++)
			{
				int currentX = x + j;
				int currentY = y + i;

				if (currentX >= 0 && currentX < width && currentY >= 0 && currentY < height)
				{
					float d = d_indepth[currentY*width + currentX];

					if (d != MINF && abs(pCC - d) > thres)
					{
						d_output[y*width + x] = 0;
						d_output[width*height + y*width + x] = 0;
						return;
					}
				}
			}
		}
	}
}

extern "C" void computeMaskEdgeMapFloat4(unsigned char* d_output, float4* d_input, float* d_indepth, float threshold, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeMaskEdgeMapFloat4Device << <gridSize, blockSize >> >(d_output, d_input, d_indepth, threshold, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Resample Float Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

inline __device__ float bilinearInterpolationFloat(float x, float y, float* d_input, unsigned int imageWidth, unsigned int imageHeight)
{
	const int2 p00 = make_int2(floor(x), floor(y));
	const int2 p01 = p00 + make_int2(0.0f, 1.0f);
	const int2 p10 = p00 + make_int2(1.0f, 0.0f);
	const int2 p11 = p00 + make_int2(1.0f, 1.0f);

	const float alpha = x - p00.x;
	const float beta = y - p00.y;

	float s0 = 0.0f; float w0 = 0.0f;
	if (p00.x < imageWidth && p00.y < imageHeight) { float v00 = d_input[p00.y*imageWidth + p00.x]; if (v00 != MINF) { s0 += (1.0f - alpha)*v00; w0 += (1.0f - alpha); } }
	if (p10.x < imageWidth && p10.y < imageHeight) { float v10 = d_input[p10.y*imageWidth + p10.x]; if (v10 != MINF) { s0 += alpha *v10; w0 += alpha; } }

	float s1 = 0.0f; float w1 = 0.0f;
	if (p01.x < imageWidth && p01.y < imageHeight) { float v01 = d_input[p01.y*imageWidth + p01.x]; if (v01 != MINF) { s1 += (1.0f - alpha)*v01; w1 += (1.0f - alpha); } }
	if (p11.x < imageWidth && p11.y < imageHeight) { float v11 = d_input[p11.y*imageWidth + p11.x]; if (v11 != MINF) { s1 += alpha *v11; w1 += alpha; } }

	const float p0 = s0 / w0;
	const float p1 = s1 / w1;

	float ss = 0.0f; float ww = 0.0f;
	if (w0 > 0.0f) { ss += (1.0f - beta)*p0; ww += (1.0f - beta); }
	if (w1 > 0.0f) { ss += beta *p1; ww += beta; }

	if (ww > 0.0f) return ss / ww;
	else		  return MINF;
}

__global__ void resampleFloatMapDevice(float* d_colorMapResampledFloat, float* d_colorMapFloat, unsigned int inputWidth, unsigned int inputHeight, unsigned int outputWidth, unsigned int outputHeight)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < outputWidth && y < outputHeight)
	{
		const float scaleWidth = (float)(inputWidth - 1) / (float)(outputWidth - 1);
		const float scaleHeight = (float)(inputHeight - 1) / (float)(outputHeight - 1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

		if (xInput < inputWidth && yInput < inputHeight)
		{
			d_colorMapResampledFloat[y*outputWidth + x] = bilinearInterpolationFloat((float)x*scaleWidth, (float)y*scaleHeight, d_colorMapFloat, inputWidth, inputHeight);
		}
	}
}

extern "C" void resampleFloatMap(float* d_colorMapResampledFloat, unsigned int outputWidth, unsigned int outputHeight, float* d_colorMapFloat, unsigned int inputWidth, unsigned int inputHeight)
{
	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleFloatMapDevice << <gridSize, blockSize >> >(d_colorMapResampledFloat, d_colorMapFloat, inputWidth, inputHeight, outputWidth, outputHeight);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Resample Float4 Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

inline __device__ float4 bilinearInterpolationFloat4(float x, float y, float4* d_input, unsigned int imageWidth, unsigned int imageHeight)
{
	const int2 p00 = make_int2(floor(x), floor(y));
	const int2 p01 = p00 + make_int2(0.0f, 1.0f);
	const int2 p10 = p00 + make_int2(1.0f, 0.0f);
	const int2 p11 = p00 + make_int2(1.0f, 1.0f);

	const float alpha = x - p00.x;
	const float beta = y - p00.y;

	//const float INVALID = 0.0f;
	const float INVALID = MINF;

	float4 s0 = make_float4(0.0f, 0.0f, 0.0f, 0.0f); float w0 = 0.0f;
	if (p00.x < imageWidth && p00.y < imageHeight) { float4 v00 = d_input[p00.y*imageWidth + p00.x]; if (v00.x != INVALID && v00.y != INVALID && v00.z != INVALID) { s0 += (1.0f - alpha)*v00; w0 += (1.0f - alpha); } }
	if (p10.x < imageWidth && p10.y < imageHeight) { float4 v10 = d_input[p10.y*imageWidth + p10.x]; if (v10.x != INVALID && v10.y != INVALID && v10.z != INVALID) { s0 += alpha *v10; w0 += alpha; } }

	float4 s1 = make_float4(0.0f, 0.0f, 0.0f, 0.0f); float w1 = 0.0f;
	if (p01.x < imageWidth && p01.y < imageHeight) { float4 v01 = d_input[p01.y*imageWidth + p01.x]; if (v01.x != INVALID && v01.y != INVALID && v01.z != INVALID) { s1 += (1.0f - alpha)*v01; w1 += (1.0f - alpha); } }
	if (p11.x < imageWidth && p11.y < imageHeight) { float4 v11 = d_input[p11.y*imageWidth + p11.x]; if (v11.x != INVALID && v11.y != INVALID && v11.z != INVALID) { s1 += alpha *v11; w1 += alpha; } }

	const float4 p0 = s0 / w0;
	const float4 p1 = s1 / w1;

	float4 ss = make_float4(0.0f, 0.0f, 0.0f, 0.0f); float ww = 0.0f;
	if (w0 > 0.0f) { ss += (1.0f - beta)*p0; ww += (1.0f - beta); }
	if (w1 > 0.0f) { ss += beta *p1; ww += beta; }

	if (ww > 0.0f) return ss / ww;
	else		  return make_float4(MINF, MINF, MINF, MINF);
}

__global__ void resampleFloat4MapDevice(float4* d_colorMapResampledFloat4, float4* d_colorMapFloat4, unsigned int inputWidth, unsigned int inputHeight, unsigned int outputWidth, unsigned int outputHeight)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < outputWidth && y < outputHeight)
	{
		const float scaleWidth = (float)(inputWidth - 1) / (float)(outputWidth - 1);
		const float scaleHeight = (float)(inputHeight - 1) / (float)(outputHeight - 1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

		if (xInput < inputWidth && yInput < inputHeight)
		{
			d_colorMapResampledFloat4[y*outputWidth + x] = bilinearInterpolationFloat4(x*scaleWidth, y*scaleHeight, d_colorMapFloat4, inputWidth, inputHeight);
		}
	}
}

extern "C" void resampleFloat4Map(float4* d_colorMapResampledFloat4, unsigned int outputWidth, unsigned int outputHeight, float4* d_colorMapFloat4, unsigned int inputWidth, unsigned int inputHeight)
{
	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleFloat4MapDevice << <gridSize, blockSize >> >(d_colorMapResampledFloat4, d_colorMapFloat4, inputWidth, inputHeight, outputWidth, outputHeight);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Resample Unsigned Char Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void resampleUCharMapDevice(unsigned char* d_MapResampled, unsigned int outputWidth, unsigned int outputHeight,
	unsigned char* d_Map, unsigned int inputWidth, unsigned int inputHeight)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < outputWidth && y < outputHeight)
	{
		const float scaleWidth = (float)(inputWidth - 1) / (float)(outputWidth - 1);
		const float scaleHeight = (float)(inputHeight - 1) / (float)(outputHeight - 1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

		if (xInput < inputWidth && yInput < inputHeight)
		{
			d_MapResampled[y*outputWidth + x] = d_Map[yInput*inputWidth + xInput];
		}
	}
}

extern "C" void resampleUCharMap(unsigned char* d_MapResampled, unsigned int outputWidth, unsigned int outputHeight,
	unsigned char* d_Map, unsigned int inputWidth, unsigned int inputHeight)
{
	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleUCharMapDevice << <gridSize, blockSize >> >(d_MapResampled, outputWidth, outputHeight, d_Map, inputWidth, inputHeight);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Convert Edge Mask to Float Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void convertEdgeMaskToFloatDevice(float* d_output, unsigned char* d_input, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	d_output[y*width + x] = min(d_input[y*width + x], d_input[width*height + y*width + x]);
}

extern "C" void convertEdgeMaskToFloat(float* d_output, unsigned char* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertEdgeMaskToFloatDevice << <gridSize, blockSize >> >(d_output, d_input, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Dilate Depth Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void dilateDepthMapDevice(float* d_output, float* d_input, float* d_inputOrig, int structureSize, int width, int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= 0 && x < width && y >= 0 && y < height)
	{
		float sum = 0.0f;
		float count = 0.0f;
		float oldDepth = d_inputOrig[y*width + x];
		if (oldDepth != MINF && oldDepth != 0)
		{
			for (int i = -structureSize; i <= structureSize; i++)
			{
				for (int j = -structureSize; j <= structureSize; j++)
				{
					if (x + j >= 0 && x + j < width && y + i >= 0 && y + i < height)
					{
						const float d = d_input[(y + i)*width + (x + j)];

						if (d != MINF && d != 0.0f && fabs(d - oldDepth) < 0.05f)
						{
							sum += d;
							count += 1.0f;
						}
					}
				}
			}
		}

		if (count > ((2 * structureSize + 1)*(2 * structureSize + 1)) / 36) d_output[y*width + x] = 1.0f;
		else			 d_output[y*width + x] = MINF;
	}
}

extern "C" void dilateDepthMapMask(float* d_output, float* d_input, float* d_inputOrig, int structureSize, int width, int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	dilateDepthMapDevice << <gridSize, blockSize >> >(d_output, d_input, d_inputOrig, structureSize, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Mean Filter Depth Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void removeDevMeanMapMaskDevice(float* d_output, float* d_input, int structureSize, int width, int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	d_output[y*width + x] = d_input[y*width + x];

	if (x >= 0 && x < width && y >= 0 && y < height)
	{
		float oldDepth = d_input[y*width + x];

		float mean = 0.0f;
		float meanSquared = 0.0f;
		float count = 0.0f;
		for (int i = -structureSize; i <= structureSize; i++)
		{
			for (int j = -structureSize; j <= structureSize; j++)
			{
				if (x + j >= 0 && x + j < width && y + i >= 0 && y + i < height)
				{
					float depth = d_input[(y + i)*width + (x + j)];
					if (depth == MINF)
					{
						depth = 8.0f;
					}

					if (depth > 0.0f)
					{
						mean += depth;
						meanSquared += depth*depth;
						count += 1.0f;
					}
				}
			}
		}

		mean /= count;
		meanSquared /= count;

		float stdDev = sqrt(meanSquared - mean*mean);

		if (fabs(oldDepth - mean) > 0.5f*stdDev)// || stdDev> 0.005f)
		{
			d_output[y*width + x] = MINF;
		}
	}
}

extern "C" void removeDevMeanMapMask(float* d_output, float* d_input, int structureSize, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	removeDevMeanMapMaskDevice << <gridSize, blockSize >> >(d_output, d_input, structureSize, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}




// Nearest neighbour
inline __device__ bool getValueNearestNeighbourNoCheck(const float2& p, const float4* inputMap, unsigned int imageWidth, unsigned int imageHeight, float4* outValue)
{
	const int u = (int)(p.x + 0.5f);
	const int v = (int)(p.y + 0.5f);

	if (u < 0 || u > imageWidth || v < 0 || v > imageHeight) return false;

	*outValue = inputMap[v*imageWidth + u];

	return true;
}

inline __device__ bool getValueNearestNeighbour(const float2& p, const float4* inputMap, unsigned int imageWidth, unsigned int imageHeight, float4* outValue)
{
	bool valid = getValueNearestNeighbourNoCheck(p, inputMap, imageWidth, imageHeight, outValue);
	return valid && (outValue->x != MINF && outValue->y != MINF && outValue->z != MINF);
}

// Nearest neighbour
inline __device__ bool getValueNearestNeighbourFloatNoCheck(const float2& p, const float* inputMap, unsigned int imageWidth, unsigned int imageHeight, float* outValue)
{
	const int u = (int)(p.x + 0.5f);
	const int v = (int)(p.y + 0.5f);

	if (u < 0 || u > imageWidth || v < 0 || v > imageHeight) return false;

	*outValue = inputMap[v*imageWidth + u];

	return true;
}

inline __device__ bool getValueNearestNeighbourFloat(const float2& p, const float* inputMap, unsigned int imageWidth, unsigned int imageHeight, float* outValue)
{
	bool valid = getValueNearestNeighbourFloatNoCheck(p, inputMap, imageWidth, imageHeight, outValue);
	return valid && (*outValue != MINF);
}

/////////////////////////////////////////////
// Compute Intensity and Derivatives
/////////////////////////////////////////////

__global__ void computeIntensityAndDerivativesDevice(float* d_intensity, unsigned int imageWidth, unsigned int imageHeight, float4* d_intensityAndDerivatives)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	const int index = y*imageWidth + x;

	if (x >= 0 && x < imageWidth && y >= 0 && y < imageHeight)
	{
		d_intensityAndDerivatives[index] = make_float4(MINF, MINF, MINF, MINF);

		if (x > 0 && x < imageWidth - 1 && y > 0 && y < imageHeight - 1)
		{
			float pos00; bool valid00 = getValueNearestNeighbourFloat(make_float2(x - 1, y - 1), d_intensity, imageWidth, imageHeight, &pos00); if (!valid00) return;
			float pos01; bool valid01 = getValueNearestNeighbourFloat(make_float2(x - 1, y - 0), d_intensity, imageWidth, imageHeight, &pos01); if (!valid01) return;
			float pos02; bool valid02 = getValueNearestNeighbourFloat(make_float2(x - 1, y + 1), d_intensity, imageWidth, imageHeight, &pos02); if (!valid02) return;

			float pos10; bool valid10 = getValueNearestNeighbourFloat(make_float2(x - 0, y - 1), d_intensity, imageWidth, imageHeight, &pos10); if (!valid10) return;
			float pos11; bool valid11 = getValueNearestNeighbourFloat(make_float2(x - 0, y - 0), d_intensity, imageWidth, imageHeight, &pos11); if (!valid11) return;
			float pos12; bool valid12 = getValueNearestNeighbourFloat(make_float2(x - 0, y + 1), d_intensity, imageWidth, imageHeight, &pos12); if (!valid12) return;

			float pos20; bool valid20 = getValueNearestNeighbourFloat(make_float2(x + 1, y - 1), d_intensity, imageWidth, imageHeight, &pos20); if (!valid20) return;
			float pos21; bool valid21 = getValueNearestNeighbourFloat(make_float2(x + 1, y - 0), d_intensity, imageWidth, imageHeight, &pos21); if (!valid21) return;
			float pos22; bool valid22 = getValueNearestNeighbourFloat(make_float2(x + 1, y + 1), d_intensity, imageWidth, imageHeight, &pos22); if (!valid22) return;

			float resU = (-1.0f)*pos00 + (1.0f)*pos20 +
				(-2.0f)*pos01 + (2.0f)*pos21 +
				(-1.0f)*pos02 + (1.0f)*pos22;
			resU /= 8.0f;

			float resV = (-1.0f)*pos00 + (-2.0f)*pos10 + (-1.0f)*pos20 +
				(1.0f)*pos02 + (2.0f)*pos12 + (1.0f)*pos22;
			resV /= 8.0f;

			d_intensityAndDerivatives[index] = make_float4(pos11, resU, resV, 1.0f);
		}
	}
}

extern "C" void computeIntensityAndDerivatives(float* d_intensity, unsigned int imageWidth, unsigned int imageHeight, float4* d_intensityAndDerivatives)
{
	const dim3 gridSize((imageWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (imageHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeIntensityAndDerivativesDevice << <gridSize, blockSize >> >(d_intensity, imageWidth, imageHeight, d_intensityAndDerivatives);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}


/////////////////////////////////////////////
// Compute grdient intensity magnitude
/////////////////////////////////////////////

__global__ void computeGradientIntensityMagnitudeDevice(float4* d_inputDU, float4* d_inputDV, unsigned int imageWidth, unsigned int imageHeight, float4* d_ouput)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	const int index = y*imageWidth + x;

	d_ouput[index] = make_float4(MINF, MINF, MINF, MINF);

	float4 DU = d_inputDU[index];
	float4 DV = d_inputDV[index];

	if (DU.x != MINF && DV.x != MINF)
	{
		float m = sqrtf(DU.x*DU.x + DV.x*DV.x);

		if (m > 0.005f)
		{
			d_ouput[index] = make_float4(m, m, m, 1.0f);
		}
	}
}

extern "C" void computeGradientIntensityMagnitude(float4* d_inputDU, float4* d_inputDV, unsigned int imageWidth, unsigned int imageHeight, float4* d_ouput)
{
	const dim3 gridSize((imageWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (imageHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeGradientIntensityMagnitudeDevice << <gridSize, blockSize >> >(d_inputDU, d_inputDV, imageWidth, imageHeight, d_ouput);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Erode Depth Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void erodeDepthMapDevice(float* d_output, float* d_input, int structureSize, int width, int height, float dThresh, float fracReq)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;


	if (x >= 0 && x < width && y >= 0 && y < height)
	{


		unsigned int count = 0;

		float oldDepth = d_input[y*width + x];
		for (int i = -structureSize; i <= structureSize; i++)
		{
			for (int j = -structureSize; j <= structureSize; j++)
			{
				if (x + j >= 0 && x + j < width && y + i >= 0 && y + i < height)
				{
					float depth = d_input[(y + i)*width + (x + j)];
					if (depth == MINF || depth == 0.0f || fabs(depth - oldDepth) > dThresh)
					{
						count++;
						//d_output[y*width+x] = MINF;
						//return;
					}
				}
			}
		}

		unsigned int sum = (2 * structureSize + 1)*(2 * structureSize + 1);
		if ((float)count / (float)sum >= fracReq) {
			d_output[y*width + x] = MINF;
		}
		else {
			d_output[y*width + x] = d_input[y*width + x];
		}
	}
}

extern "C" void erodeDepthMap(float* d_output, float* d_input, int structureSize, unsigned int width, unsigned int height, float dThresh, float fracReq)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	erodeDepthMapDevice << <gridSize, blockSize >> >(d_output, d_input, structureSize, width, height, dThresh, fracReq);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// filter annotations
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void filterAnnotations_Kernel(unsigned char* d_outputInstance, const unsigned char* d_inputInstance,
	const float* d_depth, const float* d_intensity, const unsigned char* d_instanceToIdx,
	const unsigned char* d_idxToInstance, float* d_vote,
	int structureSize, int width, int height, float sigmaD, float sigmaR, float intensityScale)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;
	const int voteoffset = (y*width + x) * MAX_NUM_LABELS_PER_SCENE;

	if (x >= 0 && x < width && y >= 0 && y < height) {
		d_outputInstance[y*width + x] = d_inputInstance[y*width + x];
		float depthCenter = d_depth[y*width + x];
		float intensityCenter = d_intensity[y*width + x];
		for (int i = -structureSize; i <= structureSize; i++) {
			for (int j = -structureSize; j <= structureSize; j++) {
				if (x + j >= 0 && x + j < width && y + i >= 0 && y + i < height)
				{
					float depth = d_depth[(y + i)*width + (x + j)];
					float intensity = d_intensity[(y + i)*width + (x + j)];
					float intensityOffset = std::abs(intensityCenter - intensity) * intensityScale; //bring intensity to approx scale of depth
					float depthOffset = 0.0f;
					if (depthCenter != MINF && depth != MINF)
						depthOffset = std::abs(depthCenter - depth);
					const float weight = gaussD(sigmaD, j, i)*gaussR(sigmaR, depthOffset)*gaussR(sigmaR, intensityOffset);
					unsigned char val = d_inputInstance[(y + i)*width + (x + j)];
					unsigned char idx = d_instanceToIdx[val];
					d_vote[voteoffset + idx] += weight;
				}
			} //j
		} //i
		float maxWeight = 0.0f; unsigned char bestVal = 0; //TODO fix this part...
		for (int i = 0; i < MAX_NUM_LABELS_PER_SCENE; i++) {
			if (d_vote[voteoffset + i] > maxWeight) {
				maxWeight = d_vote[voteoffset + i];
				bestVal = d_idxToInstance[i];
			}
		}
		d_outputInstance[y*width + x] = bestVal;
	} //in bounds of image
}

extern "C" void filterAnnotations(unsigned char* d_outputInstance, const unsigned char* d_inputInstance,
	const float* d_depth, const float* d_intensity, const unsigned char* d_instanceToIdx,
	const unsigned char* d_idxToInstance, float* d_vote,
	int structureSize, int width, int height, float sigmaD, float sigmaR, float intensityScale)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	cutilSafeCall(hipMemset(d_vote, 0, sizeof(float)*width*height*MAX_NUM_LABELS_PER_SCENE));

	filterAnnotations_Kernel << <gridSize, blockSize >> >(d_outputInstance, d_inputInstance,
		d_depth, d_intensity, d_instanceToIdx, d_idxToInstance, d_vote,
		structureSize, width, height, sigmaD, sigmaR, intensityScale);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}


__global__ void convertInstanceToLabel_Kernel(unsigned short* d_outputLabel, const unsigned char* d_inputInstance,
	const unsigned short* d_instanceToLabel, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		d_outputLabel[y*width + x] = d_instanceToLabel[d_inputInstance[y*width + x]];
	} //in bounds of image
}

extern "C" void convertInstanceToLabel(unsigned short* d_outputLabel, const unsigned char* d_inputInstance,
	const unsigned short* d_instanceToLabel, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertInstanceToLabel_Kernel << <gridSize, blockSize >> >(d_outputLabel, d_inputInstance,
		d_instanceToLabel, width, height);

	//TODO convert instance to label
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

#endif // _FILTER_
